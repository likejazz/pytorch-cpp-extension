#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>

#include <torch/torch.h>
#include <torch/extension.h>
#include <torch/library.h>

template<class T>
__global__ void add_one_kernel(const T *const input, T *const output, const int64_t N) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        output[i] = input[i] + 1;
    }
}

torch::Tensor add_one(const torch::Tensor &input) {
    auto output = torch::zeros_like(input);

    AT_DISPATCH_ALL_TYPES(
            input.scalar_type(), "add_one_kernel", [&]() {
                const auto block_size = 128;
                const auto num_blocks = std::min(
                        65535L,
                        (input.numel() + block_size - 1) / block_size
                );
                add_one_kernel<<<num_blocks, block_size>>>(
                        input.data_ptr<scalar_t>(),
                        output.data_ptr<scalar_t>(),
                        input.numel()
                );
                C10_CUDA_KERNEL_LAUNCH_CHECK();
            }
    );

    return output;
}


// @formatter:off
// CMake Interface
TORCH_LIBRARY(cudaexpr, m) {
    m.def("add_one(Tensor input) -> Tensor");
    m.impl("add_one", c10::DispatchKey::CUDA, TORCH_FN(add_one));
}

// setup.py Interface
//PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
//    m.def("add_one", &add_one);
//}
// @formatter:off